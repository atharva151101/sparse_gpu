#include "hip/hip_runtime.h"
#include "add_cuda.h"

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>
#include <sstream>


#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        throw std::runtime_error(hipGetErrorString(status));                  \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        throw std::runtime_error(hipsparseGetErrorString(status));              \
    }                                                                          \
}

template<typename value_t, typename size_t>
__global__ void add_kernel(value_t *__restrict__ r, const value_t *__restrict__ a, const value_t *__restrict__ b, const size_t n) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        r[i] = a[i] + b[i];
    }
}

float *gpu_add_f32(const float *x, const float *y, const uint64_t n) {
    float *result;
    CHECK_CUDA( hipMalloc(&result, n * sizeof(float)) );

    // Get device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int maxThreadsPerBlock = prop.maxThreadsPerBlock;

    int threadsPerBlock = maxThreadsPerBlock;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    add_kernel<float, uint64_t><<<blocksPerGrid, threadsPerBlock>>>(result, x, y, n);

    CHECK_CUDA( hipGetLastError() );

    return result;
}
