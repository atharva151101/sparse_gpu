#include "cuda_utils.h"
#include <iostream>
#include <hip/hip_runtime.h>

// Wrapper function for hipFree
void cudaFreeWrapper(void* ptr) noexcept {
    // Can't do this due to `noexcept`
    // hipError_t error = hipFree(ptr);
    // if (error != hipSuccess) {
    //     throw std::runtime_error(hipGetErrorString(error));
    // }
    hipFree(ptr);
}

void print_cuda(int * & ptr, int size) {
    int* h_ptr = new int[size];
    CHECK_CUDA(hipMemcpy(h_ptr, ptr, size*sizeof(int), hipMemcpyDeviceToHost));
    printf("\nDevice array: ");
    for(int i=0; i<size; i++) {
        printf(" %d", h_ptr[i]);
    }
    
    delete[] h_ptr;
}

